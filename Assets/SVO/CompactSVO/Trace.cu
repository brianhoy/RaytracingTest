#include "hip/hip_runtime.h"
__device__ void cast_ray(
	int* root, // In: Octree root (pointer to global mem).
	volatile float3& p, // In: Ray origin (shared mem).
	volatile float3& d, // In: Ray direction (shared mem).
	volatile float& ray_size_coef, // In: LOD at ray origin (shared mem).
	float ray_size_bias, // In: LOD increase along ray (register).
	float& hit_t, // Out: Hit t-value (register).
	float3& hit_pos, // Out: Hit position (register).
	int*& hit_parent, // Out: Hit parent voxel (pointer to global mem).
	int& hit_idx, // Out: Hit child slot index (register).
	int& hit_scale) // Out: Hit scale (register).
{
	const int s_max = 23; // Maximum scale (number of float mantissa bits).
	const float epsilon = exp2f(-s_max);
	int2 stack[s_max + 1]; // Stack of parent voxels (local mem).

	// Get rid of small ray direction components to avoid division by zero.
	if (fabsf(d.x) < epsilon) d.x = copysignf(epsilon, d.x);
	if (fabsf(d.y) < epsilon) d.y = copysignf(epsilon, d.y);
	if (fabsf(d.z) < epsilon) d.z = copysignf(epsilon, d.z);

	// Precompute the coefficients of tx(x), ty(y), and tz(z).
	// The octree is assumed to reside at coordinates [1, 2].
	float tx_coef = 1.0f / -fabs(d.x);
	float ty_coef = 1.0f / -fabs(d.y);
	float tz_coef = 1.0f / -fabs(d.z);
	float tx_bias = tx_coef * p.x;
	float ty_bias = ty_coef * p.y;
	float tz_bias = tz_coef * p.z;

	// Select octant mask to mirror the coordinate system so
	// that ray direction is negative along each axis.
	int octant_mask = 7;
	if (d.x > 0.0f) octant_mask ˆ= 1, tx_bias = 3.0f * tx_coef - tx_bias;
	if (d.y > 0.0f) octant_mask ˆ= 2, ty_bias = 3.0f * ty_coef - ty_bias;
	if (d.z > 0.0f) octant_mask ˆ= 4, tz_bias = 3.0f * tz_coef - tz_bias;

	// Initialize the active span of t-values.
	float t_min = fmaxf(fmaxf(2.0f * tx_coef - tx_bias, 2.0f * ty_coef - ty_bias), 2.0f * tz_coef - tz_bias);
	float t_max = fminf(fminf(tx_coef - tx_bias, ty_coef - ty_bias), tz_coef - tz_bias);
	float h = t_max;
	t_min = fmaxf(t_min, 0.0f);
	t_max = fminf(t_max, 1.0f);

	// Initialize the current voxel to the first child of the root.
	int* parent = root;
	int2 child_descriptor = make_int2(0, 0); // invalid until fetched
	int idx = 0;
	float3 pos = make_float3(1.0f, 1.0f, 1.0f);
	int scale = s_max - 1;
	float scale_exp2 = 0.5f; // exp2f(scale - s_max)

	if (1.5f * tx_coef - tx_bias > t_min) idx ˆ= 1, pos.x = 1.5f;
	if (1.5f * ty_coef - ty_bias > t_min) idx ˆ= 2, pos.y = 1.5f;
	if (1.5f * tz_coef - tz_bias > t_min) idx ˆ= 4, pos.z = 1.5f;

	// Traverse voxels along the ray as long as the current voxel
	// stays within the octree.
	while (scale < s_max)
	{
		// Fetch child descriptor unless it is already valid.
		if (child_descriptor.x == 0)
			child_descriptor = *(int2*)parent;

		// Determine maximum t-value of the cube by evaluating
		// tx(), ty(), and tz() at its corner.
		float tx_corner = pos.x * tx_coef - tx_bias;
		float ty_corner = pos.y * ty_coef - ty_bias;
		float tz_corner = pos.z * tz_coef - tz_bias;
		float tc_max = fminf(fminf(tx_corner, ty_corner), tz_corner);

		// Process voxel if the corresponding bit in valid mask is set
		// and the active t-span is non-empty.
		int child_shift = idx ˆ octant_mask; // permute child slots based on the mirroring
		int child_masks = child_descriptor.x << child_shift;

		if ((child_masks & 0x8000) != 0 && t_min <= t_max)
		{
			// Terminate if the voxel is small enough.
			if (tc_max * ray_size_coef + ray_size_bias >= scale_exp2)
				break; // at t_min

			// INTERSECT
			// Intersect active t-span with the cube and evaluate
			// tx(), ty(), and tz() at the center of the voxel.
			float tv_max = fminf(t_max, tc_max);
			float half = scale_exp2 * 0.5f;
			float tx_center = half * tx_coef + tx_corner;
			float ty_center = half * ty_coef + ty_corner;
			float tz_center = half * tz_coef + tz_corner;

			// Descend to the first child if the resulting t-span is non-empty.
			if (t_min <= tv_max)
			{
				// Terminate if the corresponding bit in the non-leaf mask is not set.
				if ((child_masks & 0x0080) == 0)
					break; // at t_min (overridden with tv_min).
					
				// PUSH
				// Write current parent to the stack.
				if (tc_max < h)
					stack[scale] = make_int2((int)parent, __float_as_int(t_max));
				h = tc_max;

				// Find child descriptor corresponding to the current voxel.
				int ofs = (unsigned int)child_descriptor.x >> 17; // child pointer
				if ((child_descriptor.x & 0x10000) != 0) // far
					ofs = parent[ofs * 2]; // far pointer
				ofs += popc8(child_masks & 0x7F);
				parent += ofs * 2;

				// Select child voxel that the ray enters first.
				idx = 0;
				scale--;
				scale_exp2 = half;
				if (tx_center > t_min) idx ˆ= 1, pos.x += scale_exp2;
				if (ty_center > t_min) idx ˆ= 2, pos.y += scale_exp2;
				if (tz_center > t_min) idx ˆ= 4, pos.z += scale_exp2;

				// Update active t-span and invalidate cached child descriptor.
				t_max = tv_max;
				child_descriptor.x = 0;
				continue;
			}
		}
		
		// ADVANCE
		// Step along the ray.
		int step_mask = 0;
		if (tx_corner <= tc_max) step_mask ˆ= 1, pos.x -= scale_exp2;
		if (ty_corner <= tc_max) step_mask ˆ= 2, pos.y -= scale_exp2;
		if (tz_corner <= tc_max) step_mask ˆ= 4, pos.z -= scale_exp2;

		// Update active t-span and flip bits of the child slot index.
		t_min = tc_max;
		idx ˆ= step_mask;

		// Proceed with pop if the bit flips disagree with the ray direction.
		if ((idx & step_mask) != 0)
		{
			// POP
			// Find the highest differing bit between the two positions.
			unsigned int differing_bits = 0;
			if ((step_mask & 1) != 0) differing_bits |= __float_as_int(pos.x) ˆ __float_as_int(pos.x + scale_exp2);
			if ((step_mask & 2) != 0) differing_bits |= __float_as_int(pos.y) ˆ __float_as_int(pos.y + scale_exp2);
			if ((step_mask & 4) != 0) differing_bits |= __float_as_int(pos.z) ˆ __float_as_int(pos.z + scale_exp2);
			scale = (__float_as_int((float)differing_bits) >> 23) - 127; // position of the highest bit
			scale_exp2 = __int_as_float((scale - s_max + 127) << 23); // exp2f(scale - s_max)

			// Restore parent voxel from the stack.
			int2 stackEntry = stack[scale];
			parent = (int*)stackEntry.x;
			t_max = __int_as_float(stackEntry.y);

			// Round cube position and extract child slot index.
			int shx = __float_as_int(pos.x) >> scale;
			int shy = __float_as_int(pos.y) >> scale;
			int shz = __float_as_int(pos.z) >> scale;
			pos.x = __int_as_float(shx << scale);
			pos.y = __int_as_float(shy << scale);
			pos.z = __int_as_float(shz << scale);
			idx = (shx & 1) | ((shy & 1) << 1) | ((shz & 1) << 2);

			// Prevent same parent from being stored again and invalidate cached child descriptor.
			h = 0.0f;
			child_descriptor.x = 0;
		}
	}

	// Indicate miss if we are outside the octree.
	if (scale >= s_max)
		t_min = 2.0f;

	// Undo mirroring of the coordinate system.
	if ((octant_mask & 1) == 0) pos.x = 3.0f - scale_exp2 - pos.x;
	if ((octant_mask & 2) == 0) pos.y = 3.0f - scale_exp2 - pos.y;
	if ((octant_mask & 4) == 0) pos.z = 3.0f - scale_exp2 - pos.z;

	// Output results.
	hit_t = t_min;
	hit_pos.x = fminf(fmaxf(p.x + t_min * d.x, pos.x + epsilon), pos.x + scale_exp2 - epsilon);
	hit_pos.y = fminf(fmaxf(p.y + t_min * d.y, pos.y + epsilon), pos.y + scale_exp2 - epsilon);
	hit_pos.z = fminf(fmaxf(p.z + t_min * d.z, pos.z + epsilon), pos.z + scale_exp2 - epsilon);
	hit_parent = parent;
	hit_idx = idx ˆ octant_mask ˆ 7;
	hit_scale = scale;
}
